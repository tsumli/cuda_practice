#include "hip/hip_runtime.h"
/**
 * @file main.cu
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include "common/cuda/exception.h"

__global__ void hello_device() { printf("Hello Device!\n"); }

int main() {
    std::cout << "Hello Host!" << std::endl;
    hello_device<<<1, 1>>>();
    THROW_IF_FAILED(hipDeviceSynchronize());
    return EXIT_SUCCESS;
}
