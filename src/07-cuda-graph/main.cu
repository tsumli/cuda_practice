/**
 * @file main.cu
 * @ref
 * https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#cuda-graphs
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>

#include "common/cuda/exception.h"

int main() {
    hipGraph_t graph;
    hipStream_t stream;
    THROW_IF_FAILED(hipStreamCreate(&stream));
    THROW_IF_FAILED(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    // Create the graph - it starts out empty
    THROW_IF_FAILED(hipGraphCreate(&graph, 0));

    // For the purpose of this example, we'll create
    // the nodes separately from the dependencies to
    // demonstrate that it can be done in two stages.
    // Note that dependencies can also be specified
    // at node creation.
    hipGraphAddKernelNode(&a, graph, NULL, 0, &nodeParams);
    hipGraphAddKernelNode(&b, graph, NULL, 0, &nodeParams);
    hipGraphAddKernelNode(&c, graph, NULL, 0, &nodeParams);
    hipGraphAddKernelNode(&d, graph, NULL, 0, &nodeParams);

    // Now set up dependencies on each node
    hipGraphAddDependencies(graph, &a, &b, 1);  // A->B
    hipGraphAddDependencies(graph, &a, &c, 1);  // A->C
    hipGraphAddDependencies(graph, &b, &d, 1);  // B->D
    hipGraphAddDependencies(graph, &c, &d, 1);  // C->D
}