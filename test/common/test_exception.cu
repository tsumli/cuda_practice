#include <catch2/catch_all.hpp>

#include "common/cuda/exception.h"

TEST_CASE("THROW_IF_FAILED") {
    REQUIRE_NOTHROW(THROW_IF_FAILED(hipSuccess));
    REQUIRE_THROWS(THROW_IF_FAILED(hipGetDeviceProperties(nullptr, 0)));
}